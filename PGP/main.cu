
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

__global__ void KernelVectorAbs(double *vector, int size){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = gridDim.x * blockDim.x;
  while (i < size) {
    vector[i] = (vector[i] < 0) ? -vector[i] : vector[i];
    i += offset;
  }
}

int main(){
  std::ios_base::sync_with_stdio(false);
  int size, blocks, threds;
  std::cin >> blocks >> threds >> size;
  double *vector = new double[size], *result = new double[size], *cudaVecor;
  for (int i = 0; i < size; ++i)
    std::cin >> vector[i];

  hipMalloc((void **) &cudaVecor, sizeof(double) * size);
  hipMemcpy(cudaVecor, vector, sizeof(double) * size, hipMemcpyHostToDevice);
  KernelVectorAbs<<<blocks, threds>>>(cudaVecor, size);
  hipGetLastError();
  hipMemcpy(result, cudaVecor, sizeof(double) * size, hipMemcpyDeviceToHost);

		std::cout.setf(std::ios::scientific);
  std::cout.precision(10);
		for (int i = 0; i < size; ++i)
    std::cout << result[i] << ' ';
  std::cout << '\n';

  hipFree(cudaVecor);
  delete[] vector;
}
